#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <cuda/barrier>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctime>
#include <iostream>
#include <vector>
#include <random>
#include <hip/hip_bf16.h>
#include <cassert>
#include <unistd.h>

using barrier = cuda::barrier<cuda::thread_scope_block>;

void checkCudaErrors(hipError_t error, const char* file, int line) {
  if (error != hipSuccess) {
    fprintf(
        stderr,
        "CUDA error at %s:%d: %s\n",
        file,
        line,
        hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define check(err) checkCudaErrors(err, __FILE__, __LINE__)


template <uint32_t RegCount>
__device__ void warpgroup_reg_alloc() {
    asm volatile("setmaxnreg.inc.sync.aligned.u32 %0;\n" : : "n"(RegCount));
}

__device__ static void __forceinline__
init_barrier(uint64_t* bar, int thread_count, int transaction_count) {
  uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  asm volatile(
      "mbarrier.init.shared::cta.b64 [%0], %1;\n" ::"r"(bar_ptr),
      "r"(thread_count + transaction_count) : "memory");
}

__device__ static void __forceinline__ wait_barrier(uint64_t* bar, int phase) {
  uint32_t mbar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  asm volatile(
      "{\n"
      ".reg .pred P1;\n"
      "LAB_WAIT:\n"
      "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1;\n"
      "@P1 bra.uni DONE;\n"
      "bra.uni LAB_WAIT;\n"
      "DONE:\n"
      "}\n" ::"r"(mbar_ptr),
      "r"(phase):"memory");
}

__device__ static void __forceinline__
arrive_barrier(uint64_t* bar, int count) {
  uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  asm volatile(
      "mbarrier.arrive.release.cta.shared::cta.b64 _, [%0], %1;\n" ::"r"(
          bar_ptr),
      "r"(count)
      : "memory");
}

__global__ __launch_bounds__(384) void dummy1() {
  __shared__ __align__(8) uint64_t bar, bar2;
  //__shared__ barrier bar;
  int tid = threadIdx.x;
  int wg = tid / 128;
  int wgtid = tid % 128;

  if (tid == 0) {
    init_barrier(&bar, 0, 2);
    init_barrier(&bar2, 0, 1);
  }
  __syncthreads();
  if (wg == 0) {
    int phase = 0;
    if (wgtid == 0) {
      //printf("producer %d\n", wg);
      wait_barrier(&bar, phase);
      //printf("producer %d 1 done\n", wg);
      wait_barrier(&bar, phase ^ 1);
      //arrive_barrier(&bar2, 1);
      //wait_barrier(&bar, phase ^ 1);
      //printf("producer %d 2 done\n", wg);
    }
  } else {
    int phase = 0;
    if (wgtid == 0) {
      //printf("consumer %d\n", wg);
      arrive_barrier(&bar, 1);
    }
    //asm volatile("bar.sync %0, 128;" :: "r"(wg) : "memory");
    if (wgtid < 2) {
      arrive_barrier(&bar, 1);
      //wait_barrier(&bar2, 1);
      //arrive_barrier(&bar, 1);
      //printf("consumer %d done\n", wg);
    }
  }
}

__global__ __launch_bounds__(384) void dummy() {
  __shared__ barrier bar;
  int tid = threadIdx.x;
  int wg = tid / 128;
  int wgtid = tid % 128;

  if (tid == 0) {
    init(&bar, 3);
  }
  __syncthreads();

  if (wg == 0) {
    int phase = 0;
    asm volatile("{\n//test 1\n}\n" ::: "memory");
    if (wgtid == 0) {
      bar.wait(bar.arrive());
      bar.wait(bar.arrive());
    }
    asm volatile("{\n//test 2\n}\n" ::: "memory");
  } else {
    int phase = 0;
    asm volatile("{\n//test 3\n}\n" ::: "memory");
    if (wgtid == 0) {
      bar.arrive();
      bar.arrive();
    }
    asm volatile("{\n//test 4\n}\n" ::: "memory");
  }
}

__global__ __launch_bounds__(384) void dummy() {
  __shared__ barrier bar;
  int tid = threadIdx.x;
  int wg = tid / 128;
  int wgtid = tid % 128;

  if (tid == 0) {
    init(&bar, 3);
  }
  __syncthreads();

  if (wg == 0) {
    int phase = 0;
    asm volatile("{\n//test 1\n}\n" ::: "memory");
    if (wgtid == 0) {
      bar.wait(bar.arrive());
      bar.wait(bar.arrive());
    }
    asm volatile("{\n//test 2\n}\n" ::: "memory");
  } else {
    int phase = 0;
    asm volatile("{\n//test 3\n}\n" ::: "memory");
    if (wgtid == 0) {
      bar.arrive();
      bar.arrive();
    }
    asm volatile("{\n//test 4\n}\n" ::: "memory");
  }
}

int main() {
  fprintf(stderr, "GO!\n");
  dummy<<<1, 384>>>();
  check(hipDeviceSynchronize());
  fprintf(stderr, "DONE!\n");
  return 0;
}
