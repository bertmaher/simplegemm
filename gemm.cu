#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <stdio.h>
#include <iostream>

using bf16 = __hip_bfloat16;

void checkCudaErrors(hipError_t error, const char* file, int line) {
  if (error != hipSuccess) {
    fprintf(
        stderr,
        "CUDA error at %s:%d: %s\n",
        file,
        line,
        hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
}

#define check(err) checkCudaErrors(err, __FILE__, __LINE__)

__host__ __device__ int cdiv(int m, int n) {
  return (m + n - 1) / n;
}

namespace {

template <typename T>
void tmaPrint(T s[]) {
  for (int i = 0; i < 3; i++) {
    std::cout << "  " << s[i];
  }
  std::cout << "\n";
}

__host__ static inline CUtensorMap create_tma_desc(
    bf16* gmem,
    int M,
    int N,
    int BLOCK_M,
    int BLOCK_N) {
  CUtensorMap tma_desc;
  // TODO: Check these requirements against the HW spec.
  assert(BLOCK_N >= 64);
  assert(N % 64 == 0);

  // TODO: cdiv?
  // TODO" why the 64 inner dim?
  uint64_t shape[] = {64, M, N / 64};
  uint64_t stride[] = {sizeof(bf16) * N, 64 * sizeof(bf16)};
  uint32_t box_shape[] = {64, BLOCK_M, BLOCK_N / 64};
  uint32_t box_stride[] = {1, 1, 1};

  // tmaPrint(shape);
  // tmaPrint(stride);
  // tmaPrint(box_shape);
  // tmaPrint(box_stride);

  auto result = cuTensorMapEncodeTiled(
      &tma_desc,
      CU_TENSOR_MAP_DATA_TYPE_BFLOAT16,
      3,
      gmem,
      shape,
      stride,
      box_shape,
      box_stride,
      CU_TENSOR_MAP_INTERLEAVE_NONE,
      CU_TENSOR_MAP_SWIZZLE_128B,
      CU_TENSOR_MAP_L2_PROMOTION_NONE,
      CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);

  if (result != hipSuccess) {
    fprintf(stderr, "TMA desc creation failed\n");
    exit(EXIT_FAILURE);
  }

  return tma_desc;
}

__device__ uint64_t matrix_descriptor_encode(uint64_t x) {
  return (x & 0x3ffff) >> 4;
}

__device__ uint64_t make_smem_desc(bf16* ptr) {
  constexpr uint64_t leading_dim_byte_offset = 16;
  constexpr uint64_t stride_dim_byte_offset = 1024;
  constexpr uint64_t swizzle_128b = 1ull;
  uint32_t addr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));
  return matrix_descriptor_encode(addr) |
      (matrix_descriptor_encode(leading_dim_byte_offset) << 16) |
      (matrix_descriptor_encode(stride_dim_byte_offset) << 32) |
      (swizzle_128b << 62);
}

template <int ScaleD, int ScaleA, int ScaleB, int TransA, int TransB>
__device__ __forceinline__ void wgmma256(float d[16][8], bf16* sA, bf16* sB) {
  uint64_t desc_a = make_smem_desc(&sA[0]);
  uint64_t desc_b = make_smem_desc(&sB[0]);
  // if (threadIdx.x == 128) {

  //   printf("%llx\n", desc_a);

  //   printf("%llx\n", desc_b);
  // }

#if 1
  asm volatile(
      "{\n"
      "wgmma.mma_async.sync.aligned.m64n256k16.f32.bf16.bf16 "
      "{%0,   %1,   %2,   %3,   %4,   %5,   %6,   %7,   "
      " %8,   %9,   %10,  %11,  %12,  %13,  %14,  %15,  "
      " %16,  %17,  %18,  %19,  %20,  %21,  %22,  %23,  "
      " %24,  %25,  %26,  %27,  %28,  %29,  %30,  %31,  "
      " %32,  %33,  %34,  %35,  %36,  %37,  %38,  %39,  "
      " %40,  %41,  %42,  %43,  %44,  %45,  %46,  %47,  "
      " %48,  %49,  %50,  %51,  %52,  %53,  %54,  %55,  "
      " %56,  %57,  %58,  %59,  %60,  %61,  %62,  %63,  "
      " %64,  %65,  %66,  %67,  %68,  %69,  %70,  %71,  "
      " %72,  %73,  %74,  %75,  %76,  %77,  %78,  %79,  "
      " %80,  %81,  %82,  %83,  %84,  %85,  %86,  %87,  "
      " %88,  %89,  %90,  %91,  %92,  %93,  %94,  %95,  "
      " %96,  %97,  %98,  %99,  %100, %101, %102, %103,  "
      " %104, %105, %106, %107, %108, %109, %110, %111,  "
      " %112, %113, %114, %115, %116, %117, %118, %119,  "
      " %120, %121, %122, %123, %124, %125, %126, %127},"
      " %128,"
      " %129,"
      " %130,    %131,  %132,  %133,  %134;\n"
      "}\n"
      : "+f"(d[0][0]),
        "+f"(d[0][1]),
        "+f"(d[0][2]),
        "+f"(d[0][3]),
        "+f"(d[0][4]),
        "+f"(d[0][5]),
        "+f"(d[0][6]),
        "+f"(d[0][7]),
        "+f"(d[1][0]),
        "+f"(d[1][1]),
        "+f"(d[1][2]),
        "+f"(d[1][3]),
        "+f"(d[1][4]),
        "+f"(d[1][5]),
        "+f"(d[1][6]),
        "+f"(d[1][7]),
        "+f"(d[2][0]),
        "+f"(d[2][1]),
        "+f"(d[2][2]),
        "+f"(d[2][3]),
        "+f"(d[2][4]),
        "+f"(d[2][5]),
        "+f"(d[2][6]),
        "+f"(d[2][7]),
        "+f"(d[3][0]),
        "+f"(d[3][1]),
        "+f"(d[3][2]),
        "+f"(d[3][3]),
        "+f"(d[3][4]),
        "+f"(d[3][5]),
        "+f"(d[3][6]),
        "+f"(d[3][7]),
        "+f"(d[4][0]),
        "+f"(d[4][1]),
        "+f"(d[4][2]),
        "+f"(d[4][3]),
        "+f"(d[4][4]),
        "+f"(d[4][5]),
        "+f"(d[4][6]),
        "+f"(d[4][7]),
        "+f"(d[5][0]),
        "+f"(d[5][1]),
        "+f"(d[5][2]),
        "+f"(d[5][3]),
        "+f"(d[5][4]),
        "+f"(d[5][5]),
        "+f"(d[5][6]),
        "+f"(d[5][7]),
        "+f"(d[6][0]),
        "+f"(d[6][1]),
        "+f"(d[6][2]),
        "+f"(d[6][3]),
        "+f"(d[6][4]),
        "+f"(d[6][5]),
        "+f"(d[6][6]),
        "+f"(d[6][7]),
        "+f"(d[7][0]),
        "+f"(d[7][1]),
        "+f"(d[7][2]),
        "+f"(d[7][3]),
        "+f"(d[7][4]),
        "+f"(d[7][5]),
        "+f"(d[7][6]),
        "+f"(d[7][7]),
        "+f"(d[8][0]),
        "+f"(d[8][1]),
        "+f"(d[8][2]),
        "+f"(d[8][3]),
        "+f"(d[8][4]),
        "+f"(d[8][5]),
        "+f"(d[8][6]),
        "+f"(d[8][7]),
        "+f"(d[9][0]),
        "+f"(d[9][1]),
        "+f"(d[9][2]),
        "+f"(d[9][3]),
        "+f"(d[9][4]),
        "+f"(d[9][5]),
        "+f"(d[9][6]),
        "+f"(d[9][7]),
        "+f"(d[10][0]),
        "+f"(d[10][1]),
        "+f"(d[10][2]),
        "+f"(d[10][3]),
        "+f"(d[10][4]),
        "+f"(d[10][5]),
        "+f"(d[10][6]),
        "+f"(d[10][7]),
        "+f"(d[11][0]),
        "+f"(d[11][1]),
        "+f"(d[11][2]),
        "+f"(d[11][3]),
        "+f"(d[11][4]),
        "+f"(d[11][5]),
        "+f"(d[11][6]),
        "+f"(d[11][7]),
        "+f"(d[12][0]),
        "+f"(d[12][1]),
        "+f"(d[12][2]),
        "+f"(d[12][3]),
        "+f"(d[12][4]),
        "+f"(d[12][5]),
        "+f"(d[12][6]),
        "+f"(d[12][7]),
        "+f"(d[13][0]),
        "+f"(d[13][1]),
        "+f"(d[13][2]),
        "+f"(d[13][3]),
        "+f"(d[13][4]),
        "+f"(d[13][5]),
        "+f"(d[13][6]),
        "+f"(d[13][7]),
        "+f"(d[14][0]),
        "+f"(d[14][1]),
        "+f"(d[14][2]),
        "+f"(d[14][3]),
        "+f"(d[14][4]),
        "+f"(d[14][5]),
        "+f"(d[14][6]),
        "+f"(d[14][7]),
        "+f"(d[15][0]),
        "+f"(d[15][1]),
        "+f"(d[15][2]),
        "+f"(d[15][3]),
        "+f"(d[15][4]),
        "+f"(d[15][5]),
        "+f"(d[15][6]),
        "+f"(d[15][7])
      : "l"(desc_a),
        "l"(desc_b),
        "n"(int32_t(ScaleD)),
        "n"(int32_t(ScaleA)),
        "n"(int32_t(ScaleB)),
        "n"(int32_t(TransA)),
        "n"(int32_t(TransB)));
#endif
}

__device__ void wgmma_commit_group() {
  asm volatile("wgmma.commit_group.sync.aligned;\n" ::: "memory");
}

template <int N>
__device__ void wgmma_wait_group() {
  asm volatile("wgmma.wait_group.sync.aligned %0;\n" ::"n"(N) : "memory");
}

__device__ void wgmma_fence() {
  asm volatile("wgmma.fence.sync.aligned;\n" ::: "memory");
}

template <uint32_t REGS>
__device__ static __forceinline__ void setmaxnreg_inc() {
  asm volatile("setmaxnreg.inc.sync.aligned.u32 %0;\n" : : "n"(REGS));
}

template <uint32_t REGS>
__device__ static void __forceinline__ setmaxnreg_dec() {
  asm volatile("setmaxnreg.dec.sync.aligned.u32 %0;\n" : : "n"(REGS));
}

__device__ static void __forceinline__
init_barrier(uint64_t* bar, int thread_count, int transaction_count) {
  uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  asm volatile(
      "mbarrier.init.shared::cta.b64 [%0], %1;\n" ::"r"(bar_ptr),
      "r"(thread_count + transaction_count));
}

__device__ static void __forceinline__ wait_barrier(uint64_t* bar, int phase) {
  uint32_t mbar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  asm volatile(
      "{\n"
      ".reg .pred P1;\n"
      "LAB_WAIT:\n"
      "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1;\n"
      "@P1 bra.uni DONE;\n"
      "bra.uni LAB_WAIT;\n"
      "DONE:\n"
      "}\n" ::"r"(mbar_ptr),
      "r"(phase));
}

__device__ static void __forceinline__
arrive_barrier(uint64_t* bar, int count) {
  uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  asm volatile(
      "mbarrier.arrive.release.cta.shared::cta.b64 _, [%0], %1;\n" ::"r"(
          bar_ptr),
      "r"(count)
      : "memory");
}

__device__ static void __forceinline__
expect_bytes(uint64_t* bar, uint32_t bytes) {
  uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  asm volatile(
      "mbarrier.arrive.expect_tx.shared::cta.b64 _, [%0], %1;\n" ::"r"(bar_ptr),
      "r"(bytes));
}

__device__ static void __forceinline__ tma_load(
    bf16* dst,
    void const* const src_tma_desc,
    uint64_t* bar,
    int n,
    int m) {
  uint64_t tma_ptr = reinterpret_cast<uint64_t>(src_tma_desc);
  uint32_t bar_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(bar));
  uint32_t dst_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(dst));
  asm volatile(
      "cp.async.bulk.tensor.3d.shared::cluster.global.tile.mbarrier::complete_tx::bytes"
      " [%0], [%1, {%3, %4, %5}], [%2];"
      ::
      "r"(dst_ptr),
      "l"(tma_ptr),
      "r"(bar_ptr),
      "n"(0),
      "r"(m),
      "r"(n / 64)
      : "memory");
}

__global__ void testFill(bf16* X, int M, int N, int parity) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int m_idx = idx % M;
  int n_idx = idx / M;
  if (m_idx >= M || n_idx >= N)
    return;
  if (parity < 0) {
    X[idx] = (m_idx == n_idx) ? 1.0 : 0.0;
  } else {
    X[idx] = idx;
  }

  // int v = (idx % 8 - 4);
  // //v = (v >= 0) ? v + 1 : v;
  // //X[idx] = (bf16)(v * parity);
  // X[idx] = (float)(clock() % 8) / 8.0 - 0.5;
}

hipblasHandle_t cublas_handle;
void runCublasGemmBF16(int M, int N, int K, bf16* A, bf16* B, bf16* C) {
  float alpha = 1, beta = 0;
  // C(column major) = A(row major) * B(column major)
  hipblasStatus_t status = hipblasGemmEx(
      cublas_handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      M,
      N,
      K,
      &alpha,
      A,
      HIP_R_16BF,
      K,
      B,
      HIP_R_16BF,
      K,
      &beta,
      C,
      HIP_R_16BF,
      M,
      HIPBLAS_COMPUTE_32F,
      HIPBLAS_GEMM_DEFAULT);

  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUBLAS error: %d\n", status);
    exit(EXIT_FAILURE);
  }
}

__global__ __launch_bounds__(
    1024) void naive_gemm(bf16* A, bf16* B, bf16* C, int M, int N, int K) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < M * N) {
    int m_idx = idx % M;
    int n_idx = idx / M;
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
      sum += __bfloat162float(A[m_idx * K + k]) *
          __bfloat162float(B[k + n_idx * K]);
    }
    C[m_idx + n_idx * M] = __float2bfloat16(sum);
  }
}

void run_naive_gemm(bf16* A, bf16* B, bf16* C, int M, int N, int K) {
  naive_gemm<<<cdiv(M * N, 1024), 1024>>>(A, B, C, M, N, K);
}

constexpr int BLOCK_M = 128;
constexpr int BLOCK_N = 256;
constexpr int BLOCK_K = 64;
constexpr int NUM_SMS = 132;
constexpr int STAGES = 3;
constexpr int WARPGROUP_SIZE = 128;
constexpr int WARPGROUPS = 3;
constexpr int NUM_THREADS = WARPGROUPS * WARPGROUP_SIZE;

struct SharedStorage {
  alignas(128) bf16 A[BLOCK_M * BLOCK_K * STAGES];
  alignas(128) bf16 B[BLOCK_K * BLOCK_N * STAGES];
};

__global__ __launch_bounds__(NUM_THREADS) void gemm(
    const __grid_constant__ CUtensorMap A,
    const __grid_constant__ CUtensorMap B,
    bf16* C,
    int M,
    int N,
    int K) {
  // Producer buffers for A and B.
  extern __shared__ __align__(128) uint8_t dynamic_smem[];
  SharedStorage& smem = *reinterpret_cast<SharedStorage*>(dynamic_smem);

  // Barriers.
  __shared__ __align__(8) uint64_t prod[STAGES];
  __shared__ __align__(8) uint64_t cons[STAGES];

  int tid = threadIdx.x;
  int wgid = tid / WARPGROUP_SIZE;
  int wg_tid = tid % WARPGROUP_SIZE;

  // Init barriers.
  if (tid == 0) {
    for (int i = 0; i < STAGES; i++) {
      init_barrier(&prod[i], 0, 1);
      init_barrier(&cons[i], 0, WARPGROUPS - 1);
    }
  }
  __syncthreads();

  auto m_blocks = cdiv(M, BLOCK_M);
  auto n_blocks = cdiv(N, BLOCK_N);
  auto k_blocks = cdiv(K, BLOCK_K);

  if (wgid == 0) {
    // Producer warpgroup.
    setmaxnreg_dec<40>();
    // Mainloop.

    //int m = 0, n = 0;
    if (wg_tid == 0) {
      int phase = 0;
      int stage = 0;
      for (auto bid = blockIdx.x; bid < m_blocks * n_blocks; bid += gridDim.x) {
        auto m = bid / n_blocks;
        auto n = bid % n_blocks;
        for (int k = 0; k < k_blocks ; k++) {
          // Wait for consumer.
          // TODO: stage and phase update.
          wait_barrier(&cons[stage], phase);
          // Set expect bytes for TMA.
          expect_bytes(
              &prod[stage], sizeof(bf16) * (BLOCK_M * BLOCK_K + BLOCK_K * BLOCK_N));
          // Load A.
          // TODO: use proper stage
          tma_load(&smem.A[0], &A, &prod[stage], k * BLOCK_K, m * BLOCK_M);
          // Load B.
          tma_load(&smem.B[0], &B, &prod[stage], k * BLOCK_K, n * BLOCK_N);
          phase ^= 1;
        }
      }
    }
  } else {
    // Consumer warpgroup.
    setmaxnreg_inc<232>();

    int stage = 0;
    int phase = 0;
    if (wg_tid == 0) {
      for (int i = 0; i < STAGES; i++) {
        arrive_barrier(&cons[i], 1);
      }
    }
    for (auto bid = blockIdx.x; bid < m_blocks * n_blocks; bid += gridDim.x) {
      auto m = bid / n_blocks;
      auto n = bid % n_blocks;
      float acc[16][8];
      memset(acc, 0, sizeof(acc));
      // Mainloop.
      for (int k = 0; k < k_blocks; k++) {
        // Wait for producer.
        wait_barrier(&prod[stage], phase);

        wgmma_fence();

#pragma unroll
        for (int mma_k = 0; mma_k < BLOCK_K; mma_k += 16) {
          wgmma256<1, 1, 1, 0, 0>(
              acc, &smem.A[mma_k + (wgid - 1) * 64 * 64], &smem.B[mma_k]);
        }

        wgmma_commit_group();
        wgmma_wait_group<0>();

        // Arrive at consumer.
        if (wg_tid == 0) {
          arrive_barrier(&cons[stage], 1);
        }
        phase ^= 1;
      }
      // Write back to gmem.
      auto warp = wg_tid / 32;
      auto lane = wg_tid % 32;
      auto row = warp * 16 + lane / 4;
      auto col = (wg_tid % 4) * 2;

      row += (wgid - 1) * 64;
      auto C_BLOCK = &C[m * BLOCK_M + n * BLOCK_N * M];

      //printf("%d %d %d\n", tid - 128, row, col);
      for (int inst_n = 0; inst_n < 256; inst_n += 16) {
#define Cidx(r, c) C_BLOCK[(r) + ((c) * M)]
        // clang-format off
        // printf("%d %d %d %f\n",
        //        tid,
        //        row,
        //        col,
        //        acc[n][0]);
        Cidx(row,     inst_n + col    ) = acc[inst_n / 16][0];
        Cidx(row,     inst_n + col + 1) = acc[inst_n / 16][1];
        Cidx(row + 8, inst_n + col    ) = acc[inst_n / 16][2];
        Cidx(row + 8, inst_n + col + 1) = acc[inst_n / 16][3];
        Cidx(row,     inst_n + col + 8) = acc[inst_n / 16][4];
        Cidx(row,     inst_n + col + 9) = acc[inst_n / 16][5];
        Cidx(row + 8, inst_n + col + 8) = acc[inst_n / 16][6];
        Cidx(row + 8, inst_n + col + 9) = acc[inst_n / 16][7];
        // clang-format on
      }
    }

    // auto row = (wg_tid / 32) * 2 + wg_tid / 4;
    // if (tid == 128) {
    //   for (int i = 0; i < 16; i++) {
    //     for (int j = 0; j < 8; j++) {
    //       printf("  %6.2f", acc[i][j]);
    //     }
    //     printf("\n");
    //   }
    //   printf("\n");
    // }
  }
  // __syncthreads();
  // if (tid == 128) {
  //   printf("smem.A:\n");
  //   for (int i = 0; i < BLOCK_M; i++) {
  //     for (int j = 0; j < BLOCK_K; j++) {
  //       printf("  %6.2f", __bfloat162float(smem.A[i * BLOCK_K + j]));
  //     }
  //     printf("\n");
  //   }
  //   printf("\n");
  //   printf("smem.B:\n");
  //   for (int i = 0; i < BLOCK_K; i++) {
  //     for (int j = 0; j < BLOCK_N; j++) {
  //       printf("  %6.2f", __bfloat162float(smem.B[i + j * BLOCK_K]));
  //     }
  //     printf("\n");
  //   }
  //   printf("\n");
  // }
}

void run_gemm(bf16* A, bf16* B, bf16* C, int M, int N, int K) {
  // Compute necessary shared memory for buffers.
  size_t smem_size = sizeof(SharedStorage);
  check(hipFuncSetAttribute(
      reinterpret_cast<const void*>(gemm), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));

  // Set up TMA descriptors
  auto descA = create_tma_desc(A, M, K, BLOCK_M, BLOCK_K);
  auto descB = create_tma_desc(B, N, K, BLOCK_N, BLOCK_K);

  // Launch kernel!
  gemm<<<1, NUM_THREADS, smem_size>>>(descA, descB, C, M, N, K);
  check(hipDeviceSynchronize());
  check(hipGetLastError());
}

} // namespace

int main() {
  // int m = 6 * 11 * 128;
  // int n = 6 * 12 * 128;
  // int k = 512;

  // m = k = 8;
  // n = 16;

  int m = 128;
  int n = 256;
  int k = 128;

  // m = n = k = 8192;
  int max = 16384;

  // Allocate matrices
  __hip_bfloat16* A;
  __hip_bfloat16* B;
  __hip_bfloat16* C;
  __hip_bfloat16* Cref;

  check(hipMalloc((void**)&A, sizeof(bf16) * max * max));
  check(hipMalloc((void**)&B, sizeof(bf16) * max * max));
  check(hipMalloc((void**)&C, sizeof(bf16) * max * max));
  check(hipMalloc((void**)&Cref, sizeof(bf16) * max * max));

  // Fill with test data.
  int numel = max * max;
  testFill<<<cdiv(numel, 1024), 1024>>>(A, m, k, 1);
  testFill<<<cdiv(numel, 1024), 1024>>>(B, k, n, -1);
  check(hipGetLastError());

  // Generate cuBLAS reference.
  hipblasCreate(&cublas_handle);
  runCublasGemmBF16(m, n, k, A, B, Cref);

  // Run test kernel.
  printf("about to run gemm\n");

  run_gemm(A, B, C, m, n, k);

  // Print a slab of matrix for sanity.
  bf16* hostA = (bf16*)malloc(sizeof(bf16) * numel);
  bf16* hostB = (bf16*)malloc(sizeof(bf16) * numel);
  check(hipMemcpy(hostA, A, sizeof(bf16) * m * k, hipMemcpyDeviceToHost));
  check(hipMemcpy(hostB, B, sizeof(bf16) * n * k, hipMemcpyDeviceToHost));

  for (int i = 0; i < 8; i++) {
    for (int j = 0; j < 8; j++) {
      printf("  %6.2f", __bfloat162float(hostA[i * k + j]));
    }
    printf("\n");
  }
  printf("\n");
  for (int i = 0; i < 8; i++) {
    for (int j = 0; j < 8; j++) {
      printf("  %6.2f", __bfloat162float(hostB[i + j * k]));
    }
    printf("\n");
  }
  printf("\n");

  bf16* hostM = (bf16*)malloc(sizeof(bf16) * numel);
  auto print = [&](bf16* X) {
    check(hipMemcpy(hostM, X, sizeof(bf16) * numel, hipMemcpyDeviceToHost));
    check(hipDeviceSynchronize());
    for (int i = 0; i < 8; i++) {
      for (int j = 0; j < 8; j++) {
        printf("  %6.2f", __bfloat162float(hostM[i + j * m]));
      }
      printf("\n");
    }
    printf("\n");
  };
  // print(A);
  // print(B);
  print(C);
  print(Cref);

  // Test against cuBLAS reference.
  bf16* hostC = nullptr;
  bf16* hostCref = nullptr;
  if (true) {
    hostC = (bf16*)malloc(sizeof(bf16) * m * n);
    hostCref = (bf16*)malloc(sizeof(bf16) * m * n);

    check(hipMemcpy(hostC, C, sizeof(bf16) * m * n, hipMemcpyDeviceToHost));
    check(hipMemcpy(
        hostCref, Cref, sizeof(bf16) * m * n, hipMemcpyDeviceToHost));

    for (int i = 0; i < m * n; i++) {
      float cv = __bfloat162float(hostC[i]);
      float crefv = __bfloat162float(hostCref[i]);
      if (std::abs(cv - crefv) > 1e-5) {
        fprintf(
            stderr,
            "Failed tolerance check at idx %d, C=%f, Cref=%f\n",
            i,
            cv,
            crefv);
        exit(EXIT_FAILURE);
      }
    }
  }

  // Benchmark test kernel.

  // Free resources.
  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(Cref);
  free(hostM);
  free(hostC);
  free(hostCref);
  return 0;
}
